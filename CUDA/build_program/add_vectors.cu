#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>


__global__ void vectorAdd(int *a, int *b, int* c)
{
    int i = threadIdx.x; // Create a list of threads, x represent the number of the vector that we are in
    c[i] = a[i] + b[i];

    return;
}

int main ()
{
    int a [] = {1,2,3};
    int b [] = {4,6,3};

    int c [sizeof(a) / sizeof(int)] = {0};


    int * cudaA = 0;
    int * cudaB = 0;
    int * cudaC = 0;

    // Allocate memory on the GPU
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(a));
    hipMalloc(&cudaC, sizeof(a));

    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);


    vectorAdd<<<1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);
    // Grid with 1 block, c number of threads

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    for (int i = 0; i< 3; ++i){
        std::cout<< c[i];
    }
    return;

}